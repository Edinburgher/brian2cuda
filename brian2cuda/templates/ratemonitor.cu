#include "hip/hip_runtime.h"
{% extends 'common_group.cu' %}
{# USES_VARIABLES { rate, t, _spikespace, _clock_t, _clock_dt,
                    _num_source_neurons, _source_start, _source_stop } #}

{% block define_N %}
{% endblock %}

{% block extra_maincode %}
int current_iteration = {{owner.clock.name}}.timestep[0];
static int start_offset = current_iteration;
{% endblock %}

{% block prepare_kernel_inner %}
int num_iterations = {{owner.clock.name}}.i_end;
int size_till_now = dev{{_dynamic_t}}.size();
THRUST_CHECK_ERROR(
        dev{{_dynamic_t}}.resize(num_iterations + size_till_now - start_offset)
        );
THRUST_CHECK_ERROR(
        dev{{_dynamic_rate}}.resize(num_iterations + size_till_now - start_offset)
        );
num_threads = 1;
num_blocks = 1;
{% endblock %}

{% block kernel_call %}
kernel_{{codeobj_name}}<<<num_blocks, num_threads>>>(
    current_iteration - start_offset,
    thrust::raw_pointer_cast(&(dev{{_dynamic_rate}}[0])),
    thrust::raw_pointer_cast(&(dev{{_dynamic_t}}[0])),
    ///// HOST_PARAMETERS /////
    %HOST_PARAMETERS%);

CUDA_CHECK_ERROR("kernel_{{codeobj_name}}");
{% endblock %}

{% block kernel %}
__global__ void
{% if launch_bounds %}
__launch_bounds__(1024, {{sm_multiplier}})
{% endif %}
kernel_{{codeobj_name}}(
    int32_t current_iteration,
    {% set c_type = c_data_type(variables['rate'].dtype) %}
    {{c_type}}* ratemonitor_rate,
    {% set c_type = c_data_type(variables['t'].dtype) %}
    {{c_type}}* ratemonitor_t,
    ///// DEVICE_PARAMETERS /////
    %DEVICE_PARAMETERS%
    )
{
    using namespace brian;

    ///// KERNEL_VARIABLES /////
    %KERNEL_VARIABLES%

    int num_spikes = 0;

    if (_num_spikespace-1 != _num_source_neurons)  // we have a subgroup
    {
        // TODO shouldn't this be 'i < _num_spikespace -1'?
        for (int i=0; i < _num_spikespace; i++)
        {
            const int spiking_neuron = {{_spikespace}}[i];
            if (spiking_neuron != -1)
            {
                // check if spiking neuron is in this subgroup
                if (_source_start <= spiking_neuron && spiking_neuron < _source_stop)
                    num_spikes++;
            }
            else  // end of spiking neurons
            {
                break;
            }
        }
    }
    else  // we don't have a subgroup
    {
        num_spikes = {{_spikespace}}[_num_source_neurons];
    }

    // TODO: we should be able to use {{rate}} and {{t}} here instead of passing these
    //       additional pointers. But this results in thrust::system_error illegal memory access.
    //       Don't know why... {{rate}} and ratemonitor_rate should be the same...
    ratemonitor_rate[current_iteration] = 1.0*num_spikes/{{_clock_dt}}/_num_source_neurons;
    ratemonitor_t[current_iteration] = {{_clock_t}};
}
{% endblock %}
