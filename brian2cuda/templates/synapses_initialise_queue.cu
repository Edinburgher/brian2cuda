#include "hip/hip_runtime.h"
{% macro cu_file() %}
{# USES_VARIABLES { delay } #}
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include "code_objects/{{codeobj_name}}.h"
{% set pathobj = owner.name %}

namespace {
	int num_blocks(int objects){
		return ceil(objects / (double)brian::max_threads_per_block);
	}
	int num_threads(int objects){
		return brian::max_threads_per_block;
	}
}

__global__ void _run_{{codeobj_name}}_kernel(
	unsigned int _source_N,
	unsigned int _num_blocks,
	unsigned int _num_threads_per_block,
	double _dt,
	unsigned int _syn_N,
	unsigned int num_delays,
	bool new_mode)
{
	using namespace brian;

	int tid = threadIdx.x;

	{{pathobj}}.queue->prepare(
		tid,
		_num_threads_per_block,
		_num_blocks,
		0,
		_source_N,
		_syn_N,
		num_delays,
		{{pathobj}}_size_by_pre,
		{{pathobj}}_unique_delay_size_by_pre,
		{{pathobj}}_synapses_id_by_pre,
		// TODO: delete _delay_by_pre
		{{pathobj}}_delay_by_pre,
		{{pathobj}}_unique_delay_by_pre,
		{{pathobj}}_unique_delay_start_idx_by_pre);
	{{pathobj}}.no_or_const_delay_mode = new_mode;
}

//POS(queue_id, neuron_id, neurons_N)
#define OFFSET(a, b, c)	(a*c + b)

void _run_{{pathobj}}_initialise_queue()
{
	using namespace brian;

	{# we don't use {{N}} to avoid using {{pointer_lines}} which only work inside kernels with %DEVICE_PARAMETERS% #}
	unsigned int syn_N = {{get_array_name(owner.variables['N'], access_data=False)}}[0];
	if (syn_N == 0)
		return;

	double dt = {{owner.clock.name}}.dt[0];
	unsigned int source_N = {{owner.source.N}};
	unsigned int target_N = {{owner.target.N}};

	// DENIS: TODO check speed difference when using thrust host vectors instead for easier readability and programming comfort, e.g.:
	// thrust::host_vector<int32_t> h_synapses_synaptic_sources = dev_dynamic_array_{{owner.synapses.name}}_{{owner.synapse_sources.name}}

	///////////////////////////////////
	// Create temporary host vectors //
	///////////////////////////////////

	// pre neuron IDs, post neuron IDs and delays for all synapses (sorted by synapse IDs)
	//TODO: for multiple SynapticPathways for the same Synapses object (on_pre and on_post) the following copy is identical in both pathways initialise templates
	{% if no_or_const_delay_mode %}
	// delay (on host) was potentially set in main and needs to be copied to device for later use
	dev{{_dynamic_delay}} = {{_dynamic_delay}};
	{% else %}
	// delay (on device) was set in group_variable_set_conditional and needs to be copied to host
	{{_dynamic_delay}} = dev{{_dynamic_delay}};
	{% endif %}

	// synapse IDs and delays in connectivity matrix, projected to 1D arrays of vectors
	// sorted first by pre neuron ID, then by cuda blocks (corresponding to groups of post neuron IDs)
	// the index for one pre neuron ID and block ID is: ( pre_neuron_ID * num_blocks + block_ID )

	// vectors store synapse IDs and delays for each synapse, will be sorted by delay
	thrust::host_vector<int32_t>* h_synapses_by_pre_id = new thrust::host_vector<int32_t>[num_parallel_blocks*source_N];
	{% if not no_or_const_delay_mode %}
	thrust::host_vector<unsigned int>* h_delay_by_pre_id = new thrust::host_vector<unsigned int>[num_parallel_blocks*source_N];
	{% endif %}

	//fill vectors of connectivity matrix with synapse IDs and delay IDs (in units of simulation time step)
	unsigned int max_delay = (int)({{_dynamic_delay}}[0] / dt + 0.5);
	{% if not no_or_const_delay_mode %}
	unsigned int min_delay = max_delay;
	{% endif %}
	for(int syn_id = 0; syn_id < syn_N; syn_id++)  // loop through all synapses
	{
		// pre/post_neuron_id are integers from 0 to Nsource/Ntarget (from corresponding SynapticPathway)
		// this is relevant only when using Subgroups where they might be NOT equal to the idx in their NeuronGroup
		int32_t pre_neuron_id = {{get_array_name(owner.synapse_sources, access_data=False)}}[syn_id] - {{owner.source.start}};
		int32_t post_neuron_id = {{get_array_name(owner.synapse_targets, access_data=False)}}[syn_id] - {{owner.target.start}};

		{% if not no_or_const_delay_mode %}
		unsigned int delay = (int)({{_dynamic_delay}}[syn_id] / dt + 0.5);
		if (delay > max_delay)
			max_delay = delay;
		if (delay < min_delay)
			min_delay = delay;
		{% endif %}

		unsigned int right_queue = (post_neuron_id*num_parallel_blocks)/target_N;
		unsigned int right_offset = pre_neuron_id * num_parallel_blocks + right_queue;
		h_synapses_by_pre_id[right_offset].push_back(syn_id);

		{% if not no_or_const_delay_mode %}
		h_delay_by_pre_id[right_offset].push_back(delay);
		{% endif %}
	}
	unsigned int num_delays = max_delay + 1;  // we also need a current step

	{% if no_or_const_delay_mode %}
	{{owner.name}}_delay = max_delay;
	{% else %}
	bool scalar_delay = (max_delay == min_delay);
	if (scalar_delay)
		{{owner.name}}_delay = max_delay;
	{% endif %}

	///////////////////////////////////////
	// Create arrays for device pointers //
	///////////////////////////////////////

	// TODO rename temp
	unsigned int* temp_size_by_pre_id = new unsigned int[num_parallel_blocks*source_N];
	int32_t** temp_synapses_by_pre_id = new int32_t*[num_parallel_blocks*source_N];

	{% if not no_or_const_delay_mode %}
	int num_unique_elements;
	unsigned int* temp_unique_delay_size_by_pre_id;
	unsigned int** temp_delay_by_pre_id;
	unsigned int** temp_delay_count_by_pre_id;
	unsigned int** temp_unique_delay_start_idx_by_pre_id;
	unsigned int** temp_unique_delay_by_pre_id;
	// vectors store only unique set of delays and the corresponding start index in the h_delay_by_pre_id vectors
	thrust::host_vector<unsigned int>* h_delay_count_by_pre_id;
	thrust::host_vector<unsigned int>* h_unique_delay_start_idx_by_pre_id;
	thrust::host_vector<unsigned int>* h_unique_delay_by_pre_id;
	if (!scalar_delay)
	{
		// allocate memory only if the delays are not all the same
		temp_unique_delay_size_by_pre_id = new unsigned int[num_parallel_blocks*source_N];
		temp_delay_by_pre_id = new unsigned int*[num_parallel_blocks*source_N];
		temp_delay_count_by_pre_id =  new unsigned int*[num_parallel_blocks*source_N];
		temp_unique_delay_start_idx_by_pre_id =  new unsigned int*[num_parallel_blocks*source_N];
		temp_unique_delay_by_pre_id =  new unsigned int*[num_parallel_blocks*source_N];

		h_delay_count_by_pre_id = new thrust::host_vector<unsigned int>[num_parallel_blocks*source_N];
		h_unique_delay_start_idx_by_pre_id = new thrust::host_vector<unsigned int>[num_parallel_blocks*source_N];
		h_unique_delay_by_pre_id = new thrust::host_vector<unsigned int>[num_parallel_blocks*source_N];
	}
	{% endif %}


	int size_connectivity_matrix = 0;
	//fill temp arrays with device pointers
	for(int i = 0; i < num_parallel_blocks*source_N; i++)  // loop through connectivity matrix
	{
		int num_elements = h_synapses_by_pre_id[i].size();
		size_connectivity_matrix += num_elements;
		temp_size_by_pre_id[i] = num_elements;
		if (num_elements > {{pathobj}}_max_size)
			{{pathobj}}_max_size = num_elements;

		{% if not no_or_const_delay_mode %}
		{# delay was set using Synapses object's delay attribute: `conn = Synapses(...); conn.delay = ...` #}
		if (!scalar_delay)
		{# all delays have the same value, e.g. `conn.delay = 2*ms` or because of small jitter + rounding to dt #}
		{
			// sort synapses (values) and delays (keys) by delay
			thrust::sort_by_key(
					h_delay_by_pre_id[i].begin(), 		// keys start
					h_delay_by_pre_id[i].end(), 		// keys end
					h_synapses_by_pre_id[i].begin()		// values start
					);

			// worst case: number of unique delays is num_elements
			h_unique_delay_by_pre_id[i].resize(num_elements);
			h_delay_count_by_pre_id[i].resize(num_elements);
			// TODO resize h_unique_delay_start_idx_by_pre_id after reduce_by_key and erasing h_delay_count_by_pre_id to correct size
			h_unique_delay_start_idx_by_pre_id[i].resize(num_elements);

			// create arrays of unique delays (keys) and corresponding number of occurences (values)
			thrust::pair<thrust::host_vector<unsigned int>::iterator, thrust::host_vector<unsigned int>::iterator> end_pair;
			end_pair = thrust::reduce_by_key(
					h_delay_by_pre_id[i].begin(), 		// keys start
					h_delay_by_pre_id[i].end(), 		// keys end
					thrust::make_constant_iterator(1),	// values start (each delay has count 1 before reduction)
					h_unique_delay_by_pre_id[i].begin(),  	// unique values
					h_delay_count_by_pre_id[i].begin()  	// reduced keys
					);
			thrust::host_vector<unsigned int>::iterator unique_delay_end = end_pair.first;
			thrust::host_vector<unsigned int>::iterator count_end = end_pair.second;

			// reduce count array to get start indices of unique delays in h_delay_by_pre_id (one delay for each synapse)
			thrust::host_vector<unsigned int>::iterator idx_end;
			idx_end = thrust::exclusive_scan(
					h_delay_count_by_pre_id[i].begin(),
					h_delay_count_by_pre_id[i].end(),
					h_unique_delay_start_idx_by_pre_id[i].begin()
					);

			// erase unused vector entries
			h_delay_count_by_pre_id[i].erase(count_end, h_delay_count_by_pre_id[i].end());
			h_unique_delay_by_pre_id[i].erase(unique_delay_end, h_unique_delay_by_pre_id[i].end());
			h_unique_delay_start_idx_by_pre_id[i].erase(idx_end, h_unique_delay_start_idx_by_pre_id[i].end());
			///////////////////////////////////////////////////////////

			///////////////////////////////////////////////////////////
			//// VERION FOR HAVING ONLY synapses_id_by_pre, unique_delays and delay_start_idx
			//// TODO: delete everything with ..._delay_id_by_pre, ..._delay_count_by_pre
			//
			//// worst case: number of unique delays is num_elements
			//h_unique_delay_start_idx_by_pre_id[i].resize(num_elements);
			//
			//// set the vector of indices for the original delay vector (not unique)
			//thrust::sequence(h_unique_delay_start_idx_by_pre_id[i].begin(), h_unique_delay_start_idx_by_pre_id[i].end());
			//
			//// get delays (keys) and values (indices) for first occurence of each delay value
			//thrust::pair<thrust::host_vector<unsigned int>::iterator, thrust::host_vector<unsigned int>::iterator> end_pair;
			//end_pair = thrust::unique_by_key(
			//		h_unique_delay_by_pre_id[i].begin(),  		// keys start
			//		h_unique_delay_by_pre_id[i].end(),  		// keys end
			//		h_unique_delay_start_idx_by_pre_id[i].begin() 	// values start (position in original delay array)
			//		);
			//unique_delay_end = end_pair.first;
			//idx_end = end_pair.second;
			//
			//// erase unneded vector entries
			//h_unique_delay_by_pre_id[i].erase(unique_delay_end, h_unique_delay_by_pre_id[i].end());
			//h_unique_delay_start_idx_by_pre_id[i].erase(idx_end, h_unique_delay_start_idx_by_pre_id[i].end());
			//
			///////////////////////////////////////////////////////////

			num_unique_elements = h_unique_delay_by_pre_id[i].size();
			temp_unique_delay_size_by_pre_id[i] = num_unique_elements;
			if (num_unique_elements > {{pathobj}}_max_unique_delay_size)
				{{pathobj}}_max_unique_delay_size = num_unique_elements;
		}  // end if (!scalar_delay)
		{% endif %}{# not no_or_const_delay_mode #}

		if(num_elements > 0)
		{
			hipMalloc((void**)&temp_synapses_by_pre_id[i], sizeof(int32_t)*num_elements);
			hipMemcpy(temp_synapses_by_pre_id[i],
				thrust::raw_pointer_cast(&(h_synapses_by_pre_id[i][0])),
				sizeof(int32_t)*num_elements,
				hipMemcpyHostToDevice);

			{% if not no_or_const_delay_mode %}
			if (!scalar_delay)
			{
				hipMalloc((void**)&temp_delay_by_pre_id[i], sizeof(unsigned int)*num_elements);
				hipMalloc((void**)&temp_delay_count_by_pre_id[i], sizeof(unsigned int)*num_unique_elements);
				hipMalloc((void**)&temp_unique_delay_start_idx_by_pre_id[i], sizeof(unsigned int)*num_unique_elements);
				hipMalloc((void**)&temp_unique_delay_by_pre_id[i], sizeof(unsigned int)*num_unique_elements);
				hipMemcpy(temp_delay_by_pre_id[i],
					thrust::raw_pointer_cast(&(h_delay_by_pre_id[i][0])),
					sizeof(unsigned int)*num_elements,
					hipMemcpyHostToDevice);
				hipMemcpy(temp_delay_count_by_pre_id[i],
					thrust::raw_pointer_cast(&(h_delay_count_by_pre_id[i][0])),
					sizeof(unsigned int)*num_unique_elements,
					hipMemcpyHostToDevice);
				hipMemcpy(temp_unique_delay_start_idx_by_pre_id[i],
					thrust::raw_pointer_cast(&(h_unique_delay_start_idx_by_pre_id[i][0])),
					sizeof(unsigned int)*num_unique_elements,
					hipMemcpyHostToDevice);
				hipMemcpy(temp_unique_delay_by_pre_id[i],
					thrust::raw_pointer_cast(&(h_unique_delay_by_pre_id[i][0])),
					sizeof(unsigned int)*num_unique_elements,
					hipMemcpyHostToDevice);
			}
			{% endif %}
		}
	}
	printf("INFO connectivity matrix has size %i\n", size_connectivity_matrix);


	//copy temp arrays to device
	// DENIS: TODO: rename those temp1... variables AND: why sizeof(int32_t*) and not sizeof(unsigned int*) for last 3 cpys? typo? --> CHANGED!
	unsigned int* temp;
	hipMalloc((void**)&temp, sizeof(unsigned int)*num_parallel_blocks*source_N);
	hipMemcpy(temp, temp_size_by_pre_id, sizeof(unsigned int)*num_parallel_blocks*source_N, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL({{pathobj}}_size_by_pre), &temp, sizeof(unsigned int*));
	int32_t* temp2;
	hipMalloc((void**)&temp2, sizeof(int32_t*)*num_parallel_blocks*source_N);
	hipMemcpy(temp2, temp_synapses_by_pre_id, sizeof(int32_t*)*num_parallel_blocks*source_N, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL({{pathobj}}_synapses_id_by_pre), &temp2, sizeof(int32_t**));

	{% if not no_or_const_delay_mode %}
	if (!scalar_delay)
	{
		unsigned int* temp7;
		hipMalloc((void**)&temp7, sizeof(unsigned int)*num_parallel_blocks*source_N);
		hipMemcpy(temp7, temp_unique_delay_size_by_pre_id, sizeof(unsigned int)*num_parallel_blocks*source_N, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL({{pathobj}}_unique_delay_size_by_pre), &temp7, sizeof(unsigned int*));
		unsigned int* temp3;
		hipMalloc((void**)&temp3, sizeof(unsigned int*)*num_parallel_blocks*source_N);
		hipMemcpy(temp3, temp_delay_by_pre_id, sizeof(unsigned int*)*num_parallel_blocks*source_N, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL({{pathobj}}_delay_by_pre), &temp3, sizeof(unsigned int**));
		unsigned int* temp4;
		hipMalloc((void**)&temp4, sizeof(unsigned int*)*num_parallel_blocks*source_N);
		hipMemcpy(temp4, temp_delay_count_by_pre_id, sizeof(unsigned int*)*num_parallel_blocks*source_N, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL({{pathobj}}_delay_count_by_pre), &temp4, sizeof(unsigned int**));
		unsigned int* temp5;
		hipMalloc((void**)&temp5, sizeof(unsigned int*)*num_parallel_blocks*source_N);
		hipMemcpy(temp5, temp_unique_delay_start_idx_by_pre_id, sizeof(unsigned int*)*num_parallel_blocks*source_N, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL({{pathobj}}_unique_delay_start_idx_by_pre), &temp5, sizeof(unsigned int**));
		unsigned int* temp6;
		hipMalloc((void**)&temp6, sizeof(unsigned int*)*num_parallel_blocks*source_N);
		hipMemcpy(temp6, temp_unique_delay_by_pre_id, sizeof(unsigned int*)*num_parallel_blocks*source_N, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL({{pathobj}}_unique_delay_by_pre), &temp6, sizeof(unsigned int**));
	}
	{% endif %}

	// Create circular eventspaces in no_or_const_delay_mode
	{% if not no_or_const_delay_mode %}
	if (scalar_delay)
	{% endif %}
	{
		{% set eventspace_variable = owner.variables[owner.eventspace_name] %}
		{% set _eventspace = get_array_name(eventspace_variable, access_data=False) %}
		unsigned int num_spikespaces = dev{{_eventspace}}.size();
		if (num_delays > num_spikespaces)
		{
			for (int i = num_spikespaces; i < num_delays; i++)
			{
				{{c_data_type(eventspace_variable.dtype)}}* new_eventspace;
				hipError_t status = hipMalloc((void**)&new_eventspace,
						sizeof({{c_data_type(eventspace_variable.dtype)}})*_num_{{_eventspace}});
				if (status != hipSuccess)
				{
					printf("ERROR while allocating momory for dev{{_eventspace}}[%i] on device: %s %s %d\n",
							i, hipGetErrorString(status), __FILE__, __LINE__);
					exit(status);
				}
				dev{{_eventspace}}.push_back(new_eventspace);
			}
		}
		// Check if we have multiple synapses per source-target pair in no_or_const_delay_mode
		if ({{owner.synapses.name}}_multiple_pre_post)
		{
			printf("WARNING Multiple synapses per source-target pair and scalar delays detected in Synapses object "
					"with name ``{{owner.synapses.name}}``. Application of synaptic effects will be "
					"serialized to avoid race conditions. Consider reformulating your "
					"model to avoid multiple synapses per source-target pair in a single Synapses object by using multiple "
					"Synapses objects instead. For scalar delays this is very likely to increase simulation "
					"performance significantly due to parallelisation of synaptic effect applications.\n");
		}
	}
	
	unsigned int num_threads = num_delays;
	if(num_threads >= max_threads_per_block)
	{
		num_threads = max_threads_per_block;
	}
    unsigned int num_blocks = 1;

    // check if we have enough ressources to call kernel with given number
    // of blocks and threads
    struct hipFuncAttributes funcAttrib;
    hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(_run_{{codeobj_name}}_kernel));
    if (num_threads > funcAttrib.maxThreadsPerBlock)
    {
        // use the max num_threads before launch failure
        num_threads = funcAttrib.maxThreadsPerBlock;
        printf("WARNING Not enough ressources available to call "
               "_run_{{codeobj_name}}_kernel "
               "with maximum possible threads per block (%u). "
               "Reducing num_threads to %u. (Kernel needs %i "
               "registers per block, %i bytes of "
               "statically-allocated shared memory per block, %i "
               "bytes of local memory per thread and a total of %i "
               "bytes of user-allocated constant memory)\n",
               max_threads_per_block, num_threads, funcAttrib.numRegs,
               funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
               funcAttrib.constSizeBytes);
    }
    else
    {
        printf("INFO _run_{{codeobj_name}}_kernel\n"
               "\t%u blocks\n"
               "\t%u threads\n"
               "\t%i registers per block\n"
               "\t%i bytes statically-allocated shared memory per block\n"
               "\t%i bytes local memory per thread\n"
               "\t%i bytes user-allocated constant memory\n"
               "",
               num_blocks, num_threads, funcAttrib.numRegs,
               funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
               funcAttrib.constSizeBytes);
    }


	_run_{{codeobj_name}}_kernel<<<num_blocks, num_threads>>>(
		source_N,
		num_parallel_blocks,
		max_threads_per_block,
		dt,
		syn_N,
		num_delays,
	{% if no_or_const_delay_mode %}
		true
	{% else %}
		scalar_delay
	{% endif %}
	);

	//delete temp arrays
	delete [] h_synapses_by_pre_id;
	delete [] temp_size_by_pre_id;
	delete [] temp_synapses_by_pre_id;

	{% if not no_or_const_delay_mode %}
	delete [] h_delay_by_pre_id;
	if (!scalar_delay)
	{
		delete [] h_delay_count_by_pre_id;
		delete [] h_unique_delay_start_idx_by_pre_id;
		delete [] h_unique_delay_by_pre_id;
		delete [] temp_unique_delay_size_by_pre_id;
		delete [] temp_delay_by_pre_id;
		delete [] temp_delay_count_by_pre_id;
		delete [] temp_unique_delay_start_idx_by_pre_id;
		delete [] temp_unique_delay_by_pre_id;
	}
	{% endif %}

	{% if no_or_const_delay_mode %}
	{{pathobj}}_scalar_delay = true;
	{% else %}
	{{pathobj}}_scalar_delay = scalar_delay;
	{% endif %}

	hipError_t status = hipGetLastError();
	if (status != hipSuccess)
	{
		printf("ERROR initialising {{pathobj}} in %s:%d %s\n",
				__FILE__, __LINE__, hipGetErrorString(status));
		_dealloc_arrays();
		exit(status);
	}
}

{% endmacro %}

{% macro h_file() %}
#ifndef _INCLUDED_{{codeobj_name}}
#define _INCLUDED_{{codeobj_name}}

void _run_{{codeobj_name}}();

#endif
{% endmacro %}
