#include "hip/hip_runtime.h"
{% macro cu_file() %}
#include "code_objects/{{codeobj_name}}.h"
#include "brianlib/common_math.h"
#include "brianlib/cuda_utils.h"
#include "brianlib/stdint_compat.h"
#include <cmath>
#include <stdint.h>
#include <ctime>
#include <stdio.h>
{% block extra_headers %}
{% endblock %}

////// SUPPORT CODE ///////
namespace {
    {% block random_functions %}
    // Implement dummy functions such that the host compiled code of binomial
    // functions works. Hacky, hacky ...
    double host_rand(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
    double host_randn(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
    {% endblock random_functions %}

    {% block extra_device_helper %}
    {% endblock %}

    {{support_code_lines|autoindent}}
}

{{hashdefine_lines|autoindent}}

{% block kernel %}
__global__ void
{% if launch_bounds %}
__launch_bounds__(1024, {{sm_multiplier}})
{% endif %}
kernel_{{codeobj_name}}(
    int _N,
    int THREADS_PER_BLOCK,
    ///// KERNEL_PARAMETERS /////
    %KERNEL_PARAMETERS%
    )
{
    {# USES_VARIABLES { N } #}
    using namespace brian;

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int _idx = bid * THREADS_PER_BLOCK + tid;
    int _vectorisation_idx = _idx;

    ///// KERNEL_CONSTANTS /////
    %KERNEL_CONSTANTS%

    ///// kernel_lines /////
    {{kernel_lines|autoindent}}

    assert(THREADS_PER_BLOCK == blockDim.x);

    {% block additional_variables %}
    {% endblock %}

    {% block num_thread_check %}
    if(_idx >= _N)
    {
        return;
    }
    {% endblock %}

    {% block kernel_maincode %}

    ///// scalar_code /////
    {{scalar_code|autoindent}}

    {
        ///// vector_code /////
        {{vector_code|autoindent}}

        {% block extra_vector_code %}
        {% endblock %}
    }
    {% endblock kernel_maincode %}
}
{% endblock kernel %}

void _run_{{codeobj_name}}()
{
    {# USES_VARIABLES { N } #}
    using namespace brian;

    {% block profiling_start %}
    {% if profiled %}
    const std::clock_t _start_time = std::clock();
    {% endif %}
    {% endblock %}

    {% block define_N %}
    {# N is a constant in most cases (NeuronGroup, etc.), but a scalar array for
       synapses, we therefore have to take care to get its value in the right
       way. #}
    const int _N = {{constant_or_scalar('N', variables['N'])}};
    {% endblock %}

    ///// HOST_CONSTANTS ///////////
    %HOST_CONSTANTS%

    {% block host_maincode %}
    {% endblock %}

    {% block prepare_kernel %}
    static int num_threads, num_blocks;
    static bool first_run = true;
    if (first_run)
    {
        {% block prepare_kernel_inner %}
        // get number of blocks and threads
        {% if calc_occupancy %}
        int min_num_threads; // The minimum grid size needed to achieve the
                             // maximum occupancy for a full device launch

        CUDA_SAFE_CALL(
                hipOccupancyMaxPotentialBlockSize(&min_num_threads, &num_threads,
                    kernel_{{codeobj_name}}, 0, 0)  // last args: dynamicSMemSize, blockSizeLimit
                );

        // Round up according to array size
        num_blocks = (_N + num_threads - 1) / num_threads;
        {% else %}
        num_blocks = num_parallel_blocks;
        while(num_blocks * max_threads_per_block < _N)
        {
            num_blocks *= 2;
        }
        num_threads = min(max_threads_per_block, (int)ceil(_N/(double)num_blocks));
        {% endif %}

        {% block modify_kernel_dimensions %}
        {% endblock %}

        {% endblock prepare_kernel_inner %}

        {% block occupancy %}
        // calculate theoretical occupancy
        int max_active_blocks;
        CUDA_SAFE_CALL(
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                    kernel_{{codeobj_name}}, num_threads, 0)
                );

        float occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                          (float)(max_threads_per_sm / num_threads_per_warp);
        {% endblock occupancy %}


        // check if we have enough ressources to call kernel with given number
        // of blocks and threads (can only occur for the else case above as for the
        // first max. occupancy)
        struct hipFuncAttributes funcAttrib;
        CUDA_SAFE_CALL(
                hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(kernel_{{codeobj_name}}))
                );
        if (num_threads > funcAttrib.maxThreadsPerBlock)
        {
            // use the max num_threads before launch failure
            num_threads = funcAttrib.maxThreadsPerBlock;
            printf("WARNING Not enough ressources available to call "
                   "kernel_{{codeobj_name}} "
                   "with maximum possible threads per block (%u). "
                   "Reducing num_threads to %u. (Kernel needs %i "
                   "registers per block, %i bytes of "
                   "statically-allocated shared memory per block, %i "
                   "bytes of local memory per thread and a total of %i "
                   "bytes of user-allocated constant memory)\n",
                   max_threads_per_block, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes);

            {% block update_occupancy %}
            // calculate theoretical occupancy for new num_threads
            CUDA_SAFE_CALL(
                    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                        kernel_{{codeobj_name}}, num_threads, 0)
                    );

            occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                        (float)(max_threads_per_sm / num_threads_per_warp);
            {% endblock update_occupancy %}
        }
        {% block extra_info_msg %}
        {% endblock %}
        {% block kernel_info %}
        else
        {
            printf("INFO kernel_{{codeobj_name}}\n"
                   "\t%u blocks\n"
                   "\t%u threads\n"
                   "\t%i registers per block\n"
                   "\t%i bytes statically-allocated shared memory per block\n"
                   "\t%i bytes local memory per thread\n"
                   "\t%i bytes user-allocated constant memory\n"
                   {% if calc_occupancy %}
                   "\t%.3f theoretical occupancy\n",
                   {% else %}
                   "",
                   {% endif %}
                   num_blocks, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes{% if calc_occupancy %}, occupancy{% endif %});
        }
        {% endblock %}
        first_run = false;
    }
    {% endblock prepare_kernel %}

    {% block extra_kernel_call %}
    {% endblock %}

    {% block kernel_call %}
    kernel_{{codeobj_name}}<<<num_blocks, num_threads>>>(
            _N,
            num_threads,
            ///// HOST_PARAMETERS /////
            %HOST_PARAMETERS%
        );

    CUDA_CHECK_ERROR("kernel_{{codeobj_name}}");
    {% endblock kernel_call %}

    {% block extra_kernel_call_post %}
    {% endblock %}

    {% block profiling_stop %}
    {% if profiled %}
    CUDA_SAFE_CALL(
            hipDeviceSynchronize()
            );
    const double _run_time = (double)(std::clock() -_start_time)/CLOCKS_PER_SEC;
    {{codeobj_name}}_profiling_info += _run_time;
    {% endif %}
    {% endblock %}
}

{% block extra_functions_cu %}
{% endblock %}

{% endmacro %}


{% macro h_file() %}
#ifndef _INCLUDED_{{codeobj_name}}
#define _INCLUDED_{{codeobj_name}}

#include "objects.h"

void _run_{{codeobj_name}}();

{% block extra_functions_h %}
{% endblock %}

#endif
{% endmacro %}
