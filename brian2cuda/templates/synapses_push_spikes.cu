#include "hip/hip_runtime.h"
{# USES_VARIABLES { N } #}
{% extends 'common_group.cu' %}
{# Get the name of the array that stores these events (e.g. the spikespace array) #}
{% set _eventspace = get_array_name(eventspace_variable, access_data=False) %}
#include "code_objects/{{codeobj_name}}.h"
#include "brianlib/common_math.h"
#include "brianlib/stdint_compat.h"
#include "brianlib/cuda_utils.h"
#include <assert.h>
#include <math.h>
#include <stdint.h>
#include <ctime>


{% block extra_device_helper %}
__global__ void _advance_kernel_{{codeobj_name}}()
{
    using namespace brian;
    int tid = threadIdx.x;
    {{owner.name}}.queue->advance(
        tid);
}
{% endblock extra_device_helper %}


{% block kernel %}
__global__ void
{% if launch_bounds or syn_launch_bounds %}
__launch_bounds__(1024, {{sm_multiplier}})
{% endif %}
_run_kernel_{{codeobj_name}}(
    int num_parallel_blocks,
    int _num_blocks,
    int _num_threads,
    int32_t* _eventspace)
{
    // apperently this is not always true and that is why _num_threads is passed as function argument
    // if this assert never fails, we could remove the _num_threads form the argument list
    assert(blockDim.x == _num_threads);

    using namespace brian;

    {% if not bundle_mode %}
    // TODO: check if static shared memory is faster / makes any difference
    extern __shared__ char shared_mem[];
    {% endif %}
    int bid = blockIdx.x;
    int tid = threadIdx.x;

    int post_neuron_bid = bid % num_parallel_blocks;
    int pre_neuron_idx = bid / num_parallel_blocks;

    int32_t spiking_neuron = _eventspace[pre_neuron_idx];
    assert(spiking_neuron != -1);

    // push to spikequeue if spiking_neuron is in sources of current SynapticPathway
    if({{owner.name}}.spikes_start <= spiking_neuron && spiking_neuron < {{owner.name}}.spikes_stop)
    {
        {% if bundle_mode %}
        {{owner.name}}.queue->push_bundles(
        {% else %}
        {{owner.name}}.queue->push_synapses(
            shared_mem,
        {% endif %}
            post_neuron_bid,
            tid,
            _num_threads,
            spiking_neuron - {{owner.name}}.spikes_start);
    }
}
{% endblock kernel %}


{% block define_N %}
{% endblock %}


void _run_{{codeobj_name}}()
{
    using namespace brian;

    {% if profiled %}
    const std::clock_t _start_time = std::clock();
    {% endif %}

    ///// HOST_CONSTANTS /////
    %HOST_CONSTANTS%



{% block host_maincode %}
    if ({{owner.name}}_scalar_delay)
    {
        int num_eventspaces = dev{{_eventspace}}.size();
        {{owner.name}}_eventspace_idx = (current_idx{{_eventspace}} - {{owner.name}}_delay + num_eventspaces) % num_eventspaces;

        //////////////////////////////////////////////
        //// No pushing in no_or_const_delay_mode ////
        //////////////////////////////////////////////
    }
    else if ({{owner.name}}_max_size > 0)
    {

        // get the number of spiking neurons
        int32_t num_spiking_neurons;
        CUDA_SAFE_CALL(
                hipMemcpy(&num_spiking_neurons,
                    dev{{_eventspace}}[current_idx{{_eventspace}}] + _num_{{owner.event}}space - 1,
                    sizeof(int32_t), hipMemcpyDeviceToHost)
                );

        // advance spike queues
        _advance_kernel_{{codeobj_name}}<<<1, num_parallel_blocks>>>();

        CUDA_CHECK_ERROR("_advance_kernel_{{codeobj_name}}");

    {# Don't close else bracket here, close it at end of block kernel_call, such that
       block prepare_kernel and block kernel_call are executed in this else clause #}
{% endblock host_maincode %}


{% block prepare_kernel_inner %}
    {% if not bundle_mode %}
    /* We are copying next_delay_start_idx and the atomic offset (both
     * size = num_unique_delays) into shared memory. Since
     * num_unique_delays varies for different combinations of pre
     * neuron and bid, we allocate for max(num_unique_delays). And +1
     * per block for copying size_before_resize into shared memory when
     * we need to use the outer loop.
     */
    needed_shared_memory = (2 * {{owner.name}}_max_num_unique_delays + 1) * sizeof(int);
    assert (needed_shared_memory <= max_shared_mem_size);
    {% else %}{# bundle_mode #}
    needed_shared_memory = 0;
    {% endif %}{# not bundle_mode #}

    // We don't need more then max(num_synapses) threads per block.
    num_threads = {{owner.name}}_max_size;
    if (num_threads > max_threads_per_block)
    {
        num_threads = max_threads_per_block;
    }
    // num_blocks depends on num_spiking_neurons, which changes each time step
{% endblock prepare_kernel_inner %}


{% block kernel_call %}
        if (num_spiking_neurons > 0)
        {
            num_blocks = num_parallel_blocks * num_spiking_neurons;

            _run_kernel_{{codeobj_name}}<<<num_blocks, num_threads, needed_shared_memory>>>(
                    num_parallel_blocks,
                    num_blocks,
                    num_threads,
                    dev{{_eventspace}}[current_idx{{_eventspace}}]);

            CUDA_CHECK_ERROR("_run_kernel_{{codeobj_name}}");
        }
    }  // end else if ({{owner.name}}_max_size > 0) {# from block host_maincode #}
{% endblock kernel_call %}
