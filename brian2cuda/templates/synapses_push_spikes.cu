#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//// MAIN CODE /////////////////////////////////////////////////////////////

{% macro cu_file() %}
{#  Get the name of the array that stores these events (e.g. the spikespace array) #}
    {% set _eventspace = get_array_name(eventspace_variable) %}

#include "code_objects/{{codeobj_name}}.h"
#include "brianlib/common_math.h"
#include "brianlib/stdint_compat.h"
#include <assert.h>
#include <math.h>
#include <stdint.h>
#include <ctime>

namespace {
	int _num_blocks(int num_objects)
    {
		static int needed_num_block = -1;
	    if(needed_num_block == -1)
		{
			needed_num_block = brian::num_parallel_blocks;
			while(needed_num_block * brian::max_threads_per_block < num_objects)
			{
				needed_num_block *= 2;
			}
		}
		return needed_num_block;
    }

	int _num_threads(int num_objects)
    {
		static int needed_num_threads = -1;
		if(needed_num_threads == -1)
		{
			int needed_num_block = _num_blocks(num_objects);
			needed_num_threads = min(brian::max_threads_per_block, (int)ceil(num_objects/(double)needed_num_block));
		}
		return needed_num_threads;
	}
}

__global__ void _run_{{codeobj_name}}_advance_kernel()
{
	using namespace brian;
	unsigned int tid = threadIdx.x;
	{% if no_or_const_delay_mode %}
	{{owner.name}}.which_spikespace = ({{owner.name}}.which_spikespace + 1) % {{owner.name}}.queue->max_delay;
	{% else %}
	{{owner.name}}.queue->advance(
		tid);
	{% endif %}
}

__global__ void _run_{{codeobj_name}}_push_kernel(
	unsigned int neurongroup_size,
	unsigned int _num_blocks,
	unsigned int _num_threads,
	int32_t* {{_eventspace}})
{
	// apperently this is not always true and that is why _num_threads is passed as function argument
	// if this assert never fails, we could remove the _num_threads form the argument list
	assert(blockDim.x == _num_threads);

	using namespace brian;

	// TODO: check if static shared memory is faster / makes any difference 
	extern __shared__ char shared_mem[];
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	
	// TODO: no delay mode is hard coded here!
	char no_delay_mode = false;

	// loop through spiking neurons in spikespace (indices of spiking neurons, rest -1)
	for(int i = 0; i < neurongroup_size; i++)
	{
		// spiking_neuron is index in NeuronGroup
		int32_t spiking_neuron = {{_eventspace}}[i];

		if(spiking_neuron == -1) // end of spiking neurons
		{
			assert(i == {{_eventspace}}[neurongroup_size]);
			return;
		}
		// push to spikequeue if spiking_neuron is in sources of current SynapticPathway
		if({{owner.name}}.spikes_start <= spiking_neuron && spiking_neuron < {{owner.name}}.spikes_stop)
		{
			__syncthreads();
			{{owner.name}}.queue->push(
				bid,
				tid,
				_num_threads,
				spiking_neuron - {{owner.name}}.spikes_start,
				shared_mem,
				no_delay_mode);
		}
	}
}

void _run_{{codeobj_name}}()
{
	using namespace brian;

	const std::clock_t _start_time = std::clock();

    ///// CONSTANTS ///////////
	%CONSTANTS%
	///// POINTERS ////////////

	_run_{{codeobj_name}}_advance_kernel<<<1, num_parallel_blocks>>>();

	{% if not no_or_const_delay_mode %}
	// We are copying next_delay_start_idx (size = num_unique_delays) into shared memory. Since num_unique_delays
	// varies for different combinations of pre neuron and bid, we allocate for max(num_unique_delays).
	// And +1 per block for copying size_before_resize into shared memory when we need to use the outer loop.
	unsigned int needed_shared_memory = ({{owner.name}}_max_unique_delay_size + 1) * sizeof(unsigned int);
	assert (needed_shared_memory <= max_shared_mem_size);

	// We don't need more then max(num_synapses) threads per block.
	unsigned int num_threads = {{owner.name}}_max_size;
	if (num_threads > max_threads_per_block)
	{
		num_threads = max_threads_per_block;
	}
	
	_run_{{codeobj_name}}_push_kernel<<<num_parallel_blocks, num_threads, needed_shared_memory>>>(
		_num{{eventspace_variable.name}}-1,
		num_parallel_blocks,
		num_threads,
		{% set _eventspace = get_array_name(eventspace_variable, access_data=False) %}
		dev{{_eventspace}});
	{% else %}
	//No pushing in no_or_const_delay_mode
	{% endif %}

	// Profiling
	const double _run_time = (double)(std::clock() -_start_time)/CLOCKS_PER_SEC;
	{{codeobj_name}}_profiling_info += _run_time;
}
{% endmacro %}

////////////////////////////////////////////////////////////////////////////
//// HEADER FILE ///////////////////////////////////////////////////////////

{% macro h_file() %}
#ifndef _INCLUDED_{{codeobj_name}}
#define _INCLUDED_{{codeobj_name}}

#include "objects.h"

void _run_{{codeobj_name}}();

#endif
{% endmacro %}
