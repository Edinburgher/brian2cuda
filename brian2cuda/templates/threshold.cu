#include "hip/hip_runtime.h"
{% extends 'common_group.cu' %}

{# USES_VARIABLES { N } #}
{# not_refractory and lastspike are added as needed_variables in the
   Thresholder class, we cannot use the USES_VARIABLE mechanism
   conditionally
   Same goes for "eventspace" (e.g. spikespace) which depends on the type of
   event.
#}

{% block maincode %}
	{#  Get the name of the array that stores these events (e.g. the spikespace array) #}
	{% set _eventspace = get_array_name(eventspace_variable) %}

	///// scalar_code /////
	{{scalar_code|autoindent}}

	// reset eventspace
	{{_eventspace}}[_idx] = -1;

	///// vector_code /////
	{{vector_code|autoindent}}

	if (_cond)
	{
		int32_t spike_index = atomicAdd(&{{_eventspace}}[_N], 1);
		{{_eventspace}}[spike_index] = _idx;
		{% if _uses_refractory %}
		// We have to use the pointer names directly here: The condition
		// might contain references to not_refractory or lastspike and in
		// that case the names will refer to a single entry.
		{{not_refractory}}[_idx] = false;
		{{lastspike}}[_idx] = {{t}};
		{% endif %}
	}
{% endblock %}


{% block kernel_call %}
{# N is a constant in most cases (NeuronGroup, etc.), but a scalar array for
       synapses, we therefore have to take care to get its value in the right
       way. #}
const int _N = {{constant_or_scalar('N', variables['N'])}};

hipError_t status = hipGetLastError();
if (status != hipSuccess)
{
	printf("ERROR BEFORE resetting eventspace counter in %s:%d %s\n",
			__FILE__, __LINE__, hipGetErrorString(status));
	_dealloc_arrays();
	exit(status);
}
{% set _eventspace = get_array_name(eventspace_variable, access_data=False) %}
hipMemset(&(dev{{_eventspace}}[current_idx{{_eventspace}}][_N]), 0, sizeof(int32_t));

status = hipGetLastError();
if (status != hipSuccess)
{
	printf("ERROR while resetting eventspace counter in %s:%d %s\n",
			__FILE__, __LINE__, hipGetErrorString(status));
	_dealloc_arrays();
	exit(status);
}

kernel_{{codeobj_name}}<<<num_blocks(_N),num_threads(_N)>>>(
		_N,
		num_threads(_N),
		///// HOST_PARAMETERS /////
		%HOST_PARAMETERS%
	);

status = hipGetLastError();
if (status != hipSuccess)
{
	printf("ERROR launching kernel_{{codeobj_name}} in %s:%d %s\n",
			__FILE__, __LINE__, hipGetErrorString(status));
	_dealloc_arrays();
	exit(status);
}
{% endblock kernel_call %}
