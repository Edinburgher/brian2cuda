#include "hip/hip_runtime.h"
{% extends 'common_group.cu' %}

{% block extra_device_helper %}
int mem_per_thread(){
	return sizeof(bool);
}
{% endblock %}


{% block maincode %}
	{# USES_VARIABLES { t, _spikespace, N } #}

	// use one shared spike counter per block --> parallel atomicAdd on shared memory
	// and afterwards only one atomicAdd on global memory per block
	__shared__ int32_t spike_counter_block;

	// not_refractory and lastspike are added as needed_variables in the
	// Thresholder class, we cannot use the USES_VARIABLE mechanism
	// conditionally

	//// MAIN CODE ////////////
	// scalar code
	{{scalar_code|autoindent}}

	{{_spikespace}}[_idx] = -1;

	if(tid == 0)
	{
		//init spike counter per block with 0
		spike_counter_block = 0;

		if(bid == 0)
		{
			//init global spike counter with 0
			{{_spikespace}}[N] = 0;
		}
	}
	__syncthreads();

	{{vector_code|autoindent}}
	if(_cond) {
		int32_t spike_index = atomicAdd(&spike_counter_block, 1);
		{{_spikespace}}[bid * THREADS_PER_BLOCK + spike_index] = _idx;
		{% if _uses_refractory %}
		// We have to use the pointer names directly here: The condition
		// might contain references to not_refractory or lastspike and in
		// that case the names will refer to a single entry.
		{{not_refractory}}[_idx] = false;
		{{lastspike}}[_idx] = {{t}};
		{% endif %}
	}
	__syncthreads();

	if (tid == 0) {
		atomicAdd(&{{_spikespace}}[N], spike_counter_block);
	}
{% endblock %}

{% block kernel_call %}
kernel_{{codeobj_name}}<<<num_blocks(N), num_threads(N)>>>(
		num_threads(N),
		%HOST_PARAMETERS%
	);
{% endblock %}
