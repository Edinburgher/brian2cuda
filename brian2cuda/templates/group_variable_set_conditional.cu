#include "hip/hip_runtime.h"
{% macro cu_file() %}
#include "code_objects/{{codeobj_name}}.h"
#include<math.h>
#include "brianlib/common_math.h"
#include "brianlib/stdint_compat.h"
#include<stdint.h>
#include<iostream>
#include<fstream>

{% block extra_headers %}
{% endblock %}

////// SUPPORT CODE ///////
namespace {
	{{support_code_lines|autoindent}}
}

__global__ void kernel_{{codeobj_name}}(
	unsigned int _N,
	unsigned int THREADS_PER_BLOCK,
	///// DEVICE_PARAMETERS /////
	%DEVICE_PARAMETERS%
	)
{
	{# USES_VARIABLES { N } #}
	using namespace brian;

	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x;
	unsigned int _idx = bid * THREADS_PER_BLOCK + tid;
	unsigned int _vectorisation_idx = _idx;

	///// KERNEL_VARIABLES /////
	%KERNEL_VARIABLES%

	if(_idx >= _N)
	{
		return;
	}

	///// scalar_code['condition'] /////
	{{scalar_code['condition']|autoindent}}

	///// scalar_code['statement'] /////
	{{scalar_code['statement']|autoindent}}

	///// vector_code['condition'] /////

	{{vector_code['condition']|autoindent}}
	if (_cond)
	{
		///// vector_code['statement'] /////
        {{vector_code['statement']|autoindent}}
    }
}

////// HASH DEFINES ///////
{{hashdefine_lines|autoindent}}

void _run_{{codeobj_name}}()
{
    {# USES_VARIABLES { N } #}
    {# ALLOWS_SCALAR_WRITE #}
	using namespace brian;

	{# N is a constant in most cases (NeuronGroup, etc.), but a scalar array for
           synapses, we therefore have to take care to get its value in the right
           way. #}
	const int _N = {{constant_or_scalar('N', variables['N'])}};

	///// CONSTANTS ///////////
	%CONSTANTS%

	static int num_threads, num_blocks;
	static bool first_run = true;
	if (first_run)
	{
		// get number of blocks and threads
		{% if calc_occupancy %}
		int min_num_threads; // The minimum grid size needed to achieve the
							 // maximum occupancy for a full device launch

		hipOccupancyMaxPotentialBlockSize(&min_num_threads, &num_threads,
				kernel_{{codeobj_name}}, 0, 0);  // last args: dynamicSMemSize, blockSizeLimit

		// Round up according to array size
		num_blocks = (_N + num_threads - 1) / num_threads;

		// calculate theoretical occupancy
		int max_active_blocks;
		hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
				kernel_{{codeobj_name}}, num_threads, 0);

		float occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
		                  (float)(max_threads_per_sm / num_threads_per_warp);

		{% else %}
		num_blocks = num_parallel_blocks;
		while(num_blocks * max_threads_per_block < _N)
		{
			num_blocks *= 2;
		}
		num_threads = min(max_threads_per_block, (int)ceil(_N/(double)num_blocks));
		{% endif %}

		// check if we have enough ressources to call kernel with given number of blocks and threads
		struct hipFuncAttributes funcAttrib;
		hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(kernel_{{codeobj_name}}));
		if (num_threads > funcAttrib.maxThreadsPerBlock)
		{
			// use the max num_threads before launch failure
			num_threads = funcAttrib.maxThreadsPerBlock;
			printf("WARNING Not enough ressources available to call kernel_{{codeobj_name}} with "
					"maximum possible threads per block (%u). Reducing num_threads to "
					"%u. (Kernel needs %i registers per block, %i bytes of statically-allocated "
					"shared memory per block, %i bytes of local memory per thread and "
					"a total of %i bytes of user-allocated constant memory)\n",
					max_threads_per_block, num_threads, funcAttrib.numRegs, funcAttrib.sharedSizeBytes,
					funcAttrib.localSizeBytes, funcAttrib.constSizeBytes);
		}
		else
		{
			printf("INFO calling kernel_{{codeobj_name}} with %u blocks and %u threads. "
					"Kernel needs %i registers per block, %i bytes of statically-allocated "
					"shared memory per block, %i bytes of local memory per thread and "
					"a total of %i bytes of user-allocated constant memory\n",
					num_blocks, num_threads, funcAttrib.numRegs, funcAttrib.sharedSizeBytes,
					funcAttrib.localSizeBytes, funcAttrib.constSizeBytes);
		}
		first_run = false;
	}

	kernel_{{codeobj_name}}<<<num_blocks, num_threads>>>(
		_N,
		num_threads,
		///// HOST_PARAMETERS /////
		%HOST_PARAMETERS%
	);

	hipError_t status = hipGetLastError();
	if (status != hipSuccess)
	{
		printf("ERROR launching kernel_{{codeobj_name}} in %s:%d %s\n",
				__FILE__, __LINE__, hipGetErrorString(status));
		_dealloc_arrays();
		exit(status);
	}

	{% for var in variables.itervalues() %}
	{# We want to copy only those variables that were potentially modified in aboves kernel call. #}
	{% if var is not callable and var.array and not var.constant and not var.dynamic %}
	{% set varname = get_array_name(var, access_data=False) %}
	hipMemcpy({{varname}}, dev{{varname}}, sizeof({{c_data_type(var.dtype)}})*_num_{{varname}}, hipMemcpyDeviceToHost);
	{% endif %}
	{% endfor %}
}

{% block extra_functions_cu %}
{% endblock %}

{% endmacro %}


{% macro h_file() %}
#ifndef _INCLUDED_{{codeobj_name}}
#define _INCLUDED_{{codeobj_name}}

#include "objects.h"

void _run_{{codeobj_name}}();

{% block extra_functions_h %}
{% endblock %}

#endif
{% endmacro %}



