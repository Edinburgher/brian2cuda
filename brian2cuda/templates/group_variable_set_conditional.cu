#include "hip/hip_runtime.h"
{% macro cu_file() %}
#include "code_objects/{{codeobj_name}}.h"
#include<math.h>
#include "brianlib/common_math.h"
#include "brianlib/stdint_compat.h"
#include<stdint.h>
#include<iostream>
#include<fstream>

{% block extra_headers %}
{% endblock %}

////// SUPPORT CODE ///////
namespace {
	int num_blocks(int objects){
		return ceil(objects / (double)brian::max_threads_per_block);
	}
	int num_threads(int objects){
		return brian::max_threads_per_block;
	}
	{{support_code_lines|autoindent}}
}

__global__ void _kernel_{{codeobj_name}}(
	unsigned int _N,
	unsigned int THREADS_PER_BLOCK,
	///// DEVICE_PARAMETERS /////
	%DEVICE_PARAMETERS%
	)
{
	{# USES_VARIABLES { N } #}
	using namespace brian;

	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x;
	unsigned int _idx = bid * THREADS_PER_BLOCK + tid;
	unsigned int _vectorisation_idx = _idx;

	///// KERNEL_VARIABLES /////
	%KERNEL_VARIABLES%

	if(_idx < 0 || _idx >= _N)
	{
		return;
	}

	///// scalar_code['condition'] /////
	{{scalar_code['condition']|autoindent}}

	///// scalar_code['statement'] /////
	{{scalar_code['statement']|autoindent}}

	///// vector_code['condition'] /////

	{{vector_code['condition']|autoindent}}
	if (_cond)
	{
		///// vector_code['statement'] /////
        {{vector_code['statement']|autoindent}}
    }
}

////// HASH DEFINES ///////
{{hashdefine_lines|autoindent}}

void _run_{{codeobj_name}}()
{
    {# USES_VARIABLES { N } #}
    {# ALLOWS_SCALAR_WRITE #}
	using namespace brian;

	{# N is a constant in most cases (NeuronGroup, etc.), but a scalar array for
	   synapses, we therefore have to take care to get its value in the right
	   way. #}
	const int _N = {{constant_or_scalar('N', variables['N'])}};

	///// CONSTANTS /////
	%CONSTANTS%

	_kernel_{{codeobj_name}}<<<num_blocks(_N),num_threads(_N)>>>(
		_N,
		num_threads(_N),
		///// HOST_PARAMETERS /////
		%HOST_PARAMETERS%
	);

	{% for var in variables.itervalues() %}
	{# We want to copy only those variables that were potentially modified in aboves kernel call. #}
	{% if var is not callable and var.array and not var.constant and not var.dynamic %}
	{% set varname = get_array_name(var, access_data=False) %}
	hipMemcpy({{varname}}, dev{{varname}}, sizeof({{c_data_type(var.dtype)}})*_num_{{varname}}, hipMemcpyDeviceToHost);
	{% endif %}
	{% endfor %}

}

{% block extra_functions_cu %}
{% endblock %}

{% endmacro %}


{% macro h_file() %}
#ifndef _INCLUDED_{{codeobj_name}}
#define _INCLUDED_{{codeobj_name}}

#include "objects.h"

void _run_{{codeobj_name}}();

{% block extra_functions_h %}
{% endblock %}

#endif
{% endmacro %}



