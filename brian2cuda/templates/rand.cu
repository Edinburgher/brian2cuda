{% macro cu_file() %}

#include "objects.h"
#include "synapses_classes.h"
#include "brianlib/clocks.h"
#include "network.h"
#include<iostream>
#include<fstream>
#include <hiprand.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

void _run_random_number_generation()
{
	using namespace brian;

	// Get the number of needed random numbers.
	// hiprandGenerateNormal requires an even number for pseudorandom generators
	{% for co in codeobj_with_rand %}
	static unsigned int num_rand_{{co.name}} = ({{co.owner._N}} % 2 == 0) ? {{co.owner._N}} : {{co.owner._N}} + 1;
	{% endfor %}
	{% for co in codeobj_with_randn %}
	static unsigned int num_randn_{{co.name}} = ({{co.owner._N}} % 2 == 0) ? {{co.owner._N}} : {{co.owner._N}} + 1;
	{% endfor %}

	// Allocate device memory
	static bool first_run = true;
	if (first_run)
	{
		{% for co in codeobj_with_rand | sort(attribute='name') %}
		hipMalloc((void**)&dev_{{co.name}}_rand, sizeof(float)*num_rand_{{co.name}} * {{co.rand_calls}});
		hipMemcpyToSymbol(HIP_SYMBOL(_array_{{co.name}}_rand), &dev_{{co.name}}_rand, sizeof(float*));
		{% endfor %}
		{% for co in codeobj_with_randn | sort(attribute='name') %}
		hipMalloc((void**)&dev_{{co.name}}_randn, sizeof(float)*num_randn_{{co.name}} * {{co.randn_calls}});
		hipMemcpyToSymbol(HIP_SYMBOL(_array_{{co.name}}_randn), &dev_{{co.name}}_randn, sizeof(float*));
		{% endfor %}
		first_run = false;

	}

	// Generate random numbers
	{% for co in codeobj_with_rand %}
	hiprandGenerateUniform(random_float_generator, dev_{{co.name}}_rand, num_rand_{{co.name}} * {{co.rand_calls}});
	{% endfor %}
	{% for co in codeobj_with_randn %}
	hiprandGenerateNormal(random_float_generator, dev_{{co.name}}_randn, num_randn_{{co.name}} * {{co.randn_calls}}, 0, 1);
	{% endfor %}
}
{% endmacro %}

/////////////////////////////////////////////////////////////////////////////////////////////////////

{% macro h_file() %}

#ifndef _BRIAN_RAND_H
#define _BRIAN_RAND_H

#include <hiprand.h>

void _run_random_number_generation();

#endif


{% endmacro %}
