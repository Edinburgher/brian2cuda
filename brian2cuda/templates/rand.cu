#include "hip/hip_runtime.h"
{% macro cu_file() %}

#include "objects.h"
#include "rand.h"
#include "synapses_classes.h"
#include "brianlib/clocks.h"
#include "brianlib/cuda_utils.h"
#include "network.h"
#include <hiprand.h>
#include <ctime>
#include <hiprand/hiprand_kernel.h>

using namespace brian;

// TODO make this a class member function
namespace {


    {% for co in codeobj_with_binomial | sort(attribute='name') %}
    __global__ void init_curand_states_{{co.name}}(int N)
    {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < N)
        {
            // Each thread gets the same seed, a different sequence number and
            // no offset
            // TODO: different seed and 0 sequence number is much faster, with
            // less security for independent sequences
            //hiprand_init(curand_seed + idx, 0, 0,
            hiprand_init(d_curand_seed, idx, 0,
                    &d_{{co.name}}_curand_states[idx]);
        }
    }
    {% endfor %}
}


// need a function pointer for Network::add(), can't pass a pointer to a class
// method, which is of different type
void _run_random_number_buffer()
{
    // random_number_buffer is a RandomNumberBuffer instanced, declared in objects.cu
    random_number_buffer.next_time_step();
}

{#
// TODO move stuff from objects.cu
void RandomNumberBuffer::RandomNumberBuffer()
{

    // Random seeds might be overwritten in main.cu
    unsigned long long seed = time(0);

    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_curand_seed,
                sizeof(unsigned long long))
            );

    CUDA_SAFE_CALL(
            hipMemcpy(dev_curand_seed, &seed,
                sizeof(unsigned long long), hipMemcpyHostToDevice)
            );

    CUDA_SAFE_CALL(
            hipMemcpyToSymbol(HIP_SYMBOL(d_curand_seed), &dev_curand_seed,
                sizeof(unsigned long long*))
            );


    hiprandCreateGenerator(&curand_generator, {{curand_generator_type}});
    {% if curand_generator_ordering %}
    hiprandSetGeneratorOrdering(curand_generator, {{curand_generator_ordering}});
    {% endif %}
    hiprandSetPseudoRandomGeneratorSeed(curand_generator, seed);

}
#}

void RandomNumberBuffer::init()
{
    // check that we have enough memory available
    size_t free_byte;
    size_t total_byte;
    CUDA_SAFE_CALL(
            hipMemGetInfo(&free_byte, &total_byte)
            );
    size_t num_free_floats = free_byte / sizeof(randomNumber_t);

    // number of time steps each codeobject is executed during current Network::run() call
    // XXX: we are assuming here that this function is only run in the first time step of a Network::run()
    {% for co in codeobj_with_rand_or_randn | sort(attribute='name') %}
    int64_t num_steps_this_run_{{co.name}} = {{co.owner.clock.name}}.i_end - {{co.owner.clock.name}}.t[0];
    {% endfor %}

    {% for co in codeobj_with_rand | sort(attribute='name') %}
    {% if co.template_name == 'synapses' %}
    {% set N = '_array_' + co.owner.name + '_N[0]' %}
    {% else %}
    {% set N = co.owner._N %}
    {% endif %}
    // Get the number of needed random numbers per clock cycle, the generation interval, and the number generated per hiprand call.
    num_per_cycle_rand_{{co.name}} = {{N}} * {{co.rand_calls}};
    rand_floats_per_obj_{{co.name}} = floats_per_obj;
    if (floats_per_obj < num_per_cycle_rand_{{co.name}})
        rand_floats_per_obj_{{co.name}} = num_per_cycle_rand_{{co.name}};
    rand_interval_{{co.name}} = (int)(rand_floats_per_obj_{{co.name}} / num_per_cycle_rand_{{co.name}});
    num_per_gen_rand_{{co.name}} = num_per_cycle_rand_{{co.name}} * rand_interval_{{co.name}};
    idx_rand_{{co.name}} = rand_interval_{{co.name}};

    // create max as many random numbers as will be needed during the current Network.run() call
    if ((int64_t)rand_interval_{{co.name}} > num_steps_this_run_{{co.name}})
    {
        // NOTE: if the conditional is true, we can savely cast num_steps_this_run_{{co.name}} to int
        num_per_gen_rand_{{co.name}} = num_per_cycle_rand_{{co.name}} * (int)num_steps_this_run_{{co.name}};
        assert((int64_t)num_per_cycle_rand_{{co.name}} * num_steps_this_run_{{co.name}} == num_per_gen_rand_{{co.name}});
    }

    // hiprandGenerateNormal requires an even number for pseudorandom generators
    if (num_per_gen_rand_{{co.name}} % 2 != 0)
    {
        num_per_gen_rand_{{co.name}} = num_per_gen_rand_{{co.name}} + 1;
    }

    // make sure that we don't use more memory then available
    // this checks per codeobject the number of generated floats against total available floats
    while (num_free_floats < num_per_gen_rand_{{co.name}})
    {
        printf("INFO not enough memory available to generate %i random numbers for {{co.name}}, reducing the buffer size\n", num_free_floats);
        if (num_per_gen_rand_{{co.name}} < num_per_cycle_rand_{{co.name}})
        {
            if (num_free_floats < num_per_cycle_rand_{{co.name}})
            {
                printf("ERROR not enough memory to generate random numbers for {{co.name}} %s:%d\n", __FILE__, __LINE__);
                _dealloc_arrays();
                exit(1);
            }
            else
            {
                num_per_gen_rand_{{co.name}} = num_per_cycle_rand_{{co.name}};
                break;
            }
        }
        num_per_gen_rand_{{co.name}} /= 2;
    }
    printf("INFO generating %i rand every %i clock cycles for {{co.name}}\n", num_per_gen_rand_{{co.name}}, rand_interval_{{co.name}});

    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_{{co.name}}_rand_allocator, sizeof(randomNumber_t)*num_per_gen_rand_{{co.name}})
            );
    {% endfor %}


    {% for co in codeobj_with_randn | sort(attribute='name') %}
    {% if co.template_name == 'synapses' %}
    {% set N = '_array_' + co.owner.name + '_N[0]' %}
    {% else %}
    {% set N = co.owner._N %}
    {% endif %}
    // Get the number of needed random numbers per clock cycle, the generation interval, and the number generated per hiprand call.
    num_per_cycle_randn_{{co.name}} = {{N}} * {{co.randn_calls}};
    randn_floats_per_obj_{{co.name}} = floats_per_obj;
    if (floats_per_obj < num_per_cycle_randn_{{co.name}})
        randn_floats_per_obj_{{co.name}} = num_per_cycle_randn_{{co.name}};
    randn_interval_{{co.name}} = (int)(randn_floats_per_obj_{{co.name}} / num_per_cycle_randn_{{co.name}});
    num_per_gen_randn_{{co.name}} = num_per_cycle_randn_{{co.name}} * randn_interval_{{co.name}};
    idx_randn_{{co.name}} = randn_interval_{{co.name}};

    // create max as many random numbers as will be needed during the current Network.run() call
    if ((int64_t)randn_interval_{{co.name}} > num_steps_this_run_{{co.name}})
    {
        // NOTE: if the conditional is true, we can savely cast num_steps_this_run_{{co.name}} to int
        num_per_gen_randn_{{co.name}} = num_per_cycle_randn_{{co.name}} * (int)num_steps_this_run_{{co.name}};
        assert((int64_t)num_per_cycle_randn_{{co.name}} * num_steps_this_run_{{co.name}} == num_per_gen_randn_{{co.name}});
    }

    // hiprandGenerateNormal requires an even number for pseudorandom generators
    if (num_per_gen_randn_{{co.name}} % 2 != 0)
    {
        num_per_gen_randn_{{co.name}} = num_per_gen_randn_{{co.name}} + 1;
    }

    // make sure that we don't use more memory then available
    // this checks per codeobject the number of generated floats against total available floats
    while (num_free_floats < num_per_gen_randn_{{co.name}})
    {
        printf("INFO not enough memory available to generate %i random numbers for {{co.name}}, reducing the buffer size\n", num_free_floats);
        if (num_per_gen_randn_{{co.name}} < num_per_cycle_randn_{{co.name}})
        {
            if (num_free_floats < num_per_cycle_randn_{{co.name}})
            {
                printf("ERROR not enough memory to generate random numbers for {{co.name}} %s:%d\n", __FILE__, __LINE__);
                _dealloc_arrays();
                exit(1);
            }
            else
            {
                num_per_gen_randn_{{co.name}} = num_per_cycle_randn_{{co.name}};
                break;
            }
        }
        num_per_gen_randn_{{co.name}} /= 2;
    }
    printf("INFO generating %i randn every %i clock cycles for {{co.name}}\n", num_per_gen_randn_{{co.name}}, randn_interval_{{co.name}});

    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_{{co.name}}_randn_allocator, sizeof(randomNumber_t)*num_per_gen_randn_{{co.name}})
            );
    {% endfor %}

    // now check if the total number of generated floats fit into available memory
    int total_num_generated_floats = 0;
    {% for co in codeobj_with_rand %}
    total_num_generated_floats += num_per_gen_rand_{{co.name}};
    {% endfor %}
    {% for co in codeobj_with_randn %}
    total_num_generated_floats += num_per_gen_randn_{{co.name}};
    {% endfor %}
    if (num_free_floats < total_num_generated_floats)
    {
        // TODO: find a way to deal with this? E.g. looping over buffers sorted
        // by buffer size and reducing them until it fits.
        printf("MEMORY ERROR: Trying to generate more random numbers than fit "
               "into available memory. Please report this as an issue on "
               "GitHub: https://github.com/brian-team/brian2cuda/issues/new");
        _dealloc_arrays();
        exit(1);
    }

    // allocate globabl memory for hiprand device api states
    {% for co in codeobj_with_binomial | sort(attribute='name') %}
    {% if co.template_name == 'synapses' %}
    {% set N = '_array_' + co.owner.name + '_N[0]' %}
    {% else %}
    {% set N = co.owner._N %}
    {% endif %}
    CUDA_SAFE_CALL(
            hipMalloc((void**)&dev_{{co.name}}_curand_states,
                sizeof(hiprandState) * {{N}})
            );
    CUDA_SAFE_CALL(
            hipMemcpyToSymbol(HIP_SYMBOL(d_{{co.name}}_curand_states),
                &dev_{{co.name}}_curand_states, sizeof(hiprandState*))
            );
    {% endfor %}

    // set hiprand device api states
    set_curand_device_api_states();
}


void RandomNumberBuffer::set_curand_device_api_states()
{
    {% for co in codeobj_with_binomial | sort(attribute='name') %}
    {% if co.template_name == 'synapses' %}
    {% set N = '_array_' + co.owner.name + '_N[0]' %}
    {% else %}
    {% set N = co.owner._N %}
    {% endif %}
    int num_threads, num_blocks;
    num_threads = max_threads_per_block;
    num_blocks = {{N}} / max_threads_per_block + 1;
    if ({{N}} < num_threads)
        num_threads = {{N}};
    init_curand_states_{{co.name}}<<<num_blocks, num_threads>>>({{N}});
    {% endfor %}
}


void RandomNumberBuffer::set_seed(unsigned long long seed)
{
    CUDA_CHECK_ERROR("before set seed");
    CUDA_SAFE_CALL(
            hiprandSetPseudoRandomGeneratorSeed(curand_generator, seed)
            );

    CUDA_CHECK_ERROR("after set seed");
    // generator offset needs to be reset to its default (=0)
    CUDA_SAFE_CALL(
            hiprandSetGeneratorOffset(curand_generator, 0ULL)
            );

    // reinit the buffers, dt might have changed or the num_steps_this_run_{}
    // need to free memory for init() to work
    // TODO: could be solved more efficiently:
    //      have one buffer object per codeobject and check per codeobject if
    //      dt has changed or if num_steps_this_run_ was used previously to
    //      generate less random numbers! -> issue?
    CUDA_CHECK_ERROR("after offset");
    {% for co in codeobj_with_rand | sort(attribute='name') %}
    CUDA_SAFE_CALL(
            hipFree(dev_{{co.name}}_rand_allocator)
            );
    {% endfor %}

    {% for co in codeobj_with_randn | sort(attribute='name') %}
    CUDA_SAFE_CALL(
            hipFree(dev_{{co.name}}_randn_allocator)
            );
    {% endfor %}

    // don't call init() here already since the network clocks might not be set
    // up yet, call init() only once network started running
    needs_init = true;

    // set seed for hiprand device api calls
    // don't set the same seed for host api and device api random states, just in case
    seed += 1;
    CUDA_SAFE_CALL(
            hipMemcpy(dev_curand_seed, &seed,
                sizeof(unsigned long long), hipMemcpyHostToDevice)
            );

    // update hiprand device api states with new seed
    set_curand_device_api_states();
}


void RandomNumberBuffer::refill_uniform_numbers(
        randomNumber_t* dev_rand_allocator,
        randomNumber_t* &dev_rand,
        int num_per_gen_rand,
        int &idx_rand)
{
    // generate uniform distributed random numbers and reset buffer index

    {% if curand_float_type == 'float' %}
    hiprandGenerateUniform(curand_generator, dev_rand_allocator, num_per_gen_rand);
    {% else %}
    hiprandGenerateUniformDouble(curand_generator, dev_rand_allocator, num_per_gen_rand);
    {% endif %}
    // before: XXX dev_rand = &dev_rand_allocator[0];
    dev_rand = dev_rand_allocator;
    idx_rand = 1;
}


void RandomNumberBuffer::refill_normal_numbers(
        randomNumber_t* dev_randn_allocator,
        randomNumber_t* &dev_randn,
        int num_per_gen_randn,
        int &idx_randn)
{
    // generate normal distributed random numbers and reset buffer index

    {% if curand_float_type == 'float' %}
    hiprandGenerateNormal(curand_generator, dev_randn_allocator, num_per_gen_randn, 0, 1);
    {% else %}
    hiprandGenerateNormalDouble(curand_generator, dev_randn_allocator, num_per_gen_randn, 0, 1);
    {% endif %}
    // before: XXX dev_randn = &dev_randn_allocator[0];
    dev_randn = dev_randn_allocator;
    idx_randn = 1;
}


void RandomNumberBuffer::next_time_step()
{
    if (needs_init)
    {
        init();
        needs_init = false;
    }

    {% for co in codeobj_with_rand %}
    // uniform numbers for {{co.name}}
    if (idx_rand_{{co.name}} == rand_interval_{{co.name}})
    {
        refill_uniform_numbers(
                dev_{{co.name}}_rand_allocator,
                dev_{{co.name}}_rand,
                num_per_gen_rand_{{co.name}},
                idx_rand_{{co.name}});
    }
    else
    {
        // move device pointer to next numbers
        dev_{{co.name}}_rand += num_per_cycle_rand_{{co.name}};
        idx_rand_{{co.name}} += 1;
    }
    assert(dev_{{co.name}}_rand < dev_{{co.name}}_rand_allocator + num_per_gen_rand_{{co.name}});
    {% endfor %}

    {% for co in codeobj_with_randn %}
    // normal numbers for {{co.name}}
    if (idx_randn_{{co.name}} == randn_interval_{{co.name}})
    {
        refill_normal_numbers(
                dev_{{co.name}}_randn_allocator,
                dev_{{co.name}}_randn,
                num_per_gen_randn_{{co.name}},
                idx_randn_{{co.name}});
    }
    else
    {
        // move device pointer to next numbers
        dev_{{co.name}}_randn += num_per_cycle_randn_{{co.name}};
        idx_randn_{{co.name}} += 1;
    }
    if (dev_{{co.name}}_randn < dev_{{co.name}}_randn_allocator + num_per_gen_randn_{{co.name}})
        printf("dev_randn %u, dev_randn_allocator %u, num_per_gen_randn %d\n",
                dev_{{co.name}}_randn, dev_{{co.name}}_randn_allocator,
                num_per_gen_randn_{{co.name}});
    else
        printf("ERROR: dev_randn %u, dev_randn_allocator %u, num_per_gen_randn %d\n",
                dev_{{co.name}}_randn, dev_{{co.name}}_randn_allocator,
                num_per_gen_randn_{{co.name}});
    assert(dev_{{co.name}}_randn < dev_{{co.name}}_randn_allocator + num_per_gen_randn_{{co.name}});
    {% endfor %}
}
{% endmacro %}

/////////////////////////////////////////////////////////////////////////////////////////////////////

{% macro h_file() %}

#ifndef _BRIAN_RAND_H
#define _BRIAN_RAND_H

#include <hiprand.h>

void _run_random_number_buffer();

class RandomNumberBuffer
{
    bool needs_init = true;

    // how many random numbers we want to create at once (tradeoff memory usage <-> generation overhead)
    double mb_per_obj = 50;  // MB per codeobject and rand / randn
    int floats_per_obj = (mb_per_obj * 1024.0 * 1024.0) / sizeof(randomNumber_t);

    // The number of needed random numbers per clock cycle, the generation interval, and the number generated per hiprand call.
    //
    // needed random numbers per clock cycle
    // int num_per_cycle_rand_{};
    //
    // number of time steps after which buffer needs to be refilled
    // int rand_interval_{};
    //
    // buffer size
    // int num_per_gen_rand_{};
    //
    // number of time steps since last buffer refill
    // int idx_rand_{};
    //
    // maximum number of random numbers fitting given allocated memory
    // int rand_floats_per_obj_{};


    // uniform distributed random numbers (rand)

    {% for co in codeobj_with_rand %}
    // {{co.name}}
    int num_per_cycle_rand_{{co.name}};
    int rand_interval_{{co.name}};
    int num_per_gen_rand_{{co.name}};
    int idx_rand_{{co.name}};
    int rand_floats_per_obj_{{co.name}};

    {% endfor %}

    // normal distributed random numbers (randn)

    {% for co in codeobj_with_randn %}
    // {{co.name}}
    int num_per_cycle_randn_{{co.name}};
    int randn_interval_{{co.name}};
    int num_per_gen_randn_{{co.name}};
    int idx_randn_{{co.name}};
    int randn_floats_per_obj_{{co.name}};

    {% endfor %}

    void init();
    void set_curand_device_api_states();
    void refill_uniform_numbers(randomNumber_t*, randomNumber_t*&, int, int&);
    void refill_normal_numbers(randomNumber_t*, randomNumber_t*&, int, int&);

public:
    //Network();
    void next_time_step();
    void set_seed(unsigned long long);
};

#endif

{% endmacro %}
